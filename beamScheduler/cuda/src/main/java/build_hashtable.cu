
#include <hip/hip_runtime.h>
extern "C" __global__ void build_hashtable(int *R, int R_size, int *hash_table) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;

  int key = R[offset];
  int hash = key & (R_size-1);

  if (offset < R_size) {
    hash_table[hash] = key;
  }
}