
#include <hip/hip_runtime.h>
extern "C" __global__ void probe_hashtable(int *S, int S_size, int *hash_table, int ht_size,int* OUT) {//OUT and S have the same size
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = offset; i < S_size && i < ht_size; i += stride) {
    int key = S[i];
    int hash = key & (ht_size - 1);

    if (key == hash_table[hash]) {
		OUT[i] = key;
    }else {
        OUT[i] = -1;
    }
  }
}